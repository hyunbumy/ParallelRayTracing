#include "hip/hip_runtime.h"
#include "CudaRT.h"

__device__
float3 Trace(float3& rayorig, float3& raydir, CudaSphere* objects, 
             int objSize, int depth)
{
    //if (raydir.length() != 1) std::cerr << "Error " << raydir << std::endl;
    // find intersection of this ray with the sphere in the scene
    float tnear = INFINITY;
    const CudaSphere* object = GetClosestObject(rayorig, raydir, tnear,
                                                objects, objSize);
    
    // if there's no intersection return black or background color
    if (object == nullptr) return make_float3(2, 2, 2);
    float3 surfaceColor = make_float3(0,0,0); // color of the ray/surfaceof the object intersected by the ray
    float3 phit = rayorig + raydir * tnear; // point of intersection
    float3 nhit = phit - object->center; // normal at the intersection point
    nhit = normalize(nhit); // normalize normal direction
    // If the normal and the view direction are not opposite to each other
    // reverse the normal direction. That also means we are inside the sphere so set
    // the inside bool to true. Finally reverse the sign of IdotN which we want
    // positive.
    float bias = 1e-4; // add some bias to the point from which we will be tracing
    bool inside = false;
    if (dot(raydir, nhit) > 0)
    {
        nhit *= -1;
        inside = true;
    }

    if ((object->transparency > 0 || object->reflection > 0) && depth < MAX_RAY_DEPTH) {
        float facingratio = -1 * dot(raydir, nhit);
        // change the mix value to tweak the effect
        float fresneleffect = Mix(pow(1 - facingratio, 3), 1, 0.1);
        // compute reflection direction (not need to normalize because all vectors
        // are already normalized)
        float3 refldir = raydir - nhit * 2 * dot(raydir, nhit);
        refldir = normalize(refldir);
        auto reflOrig = phit + nhit * bias;
        float3 reflection = Trace(reflOrig, refldir, objects, objSize, depth + 1);
        float3 refraction = make_float3(0,0,0);
        // if the sphere is also transparent compute refraction ray (transmission)
        if (object->transparency) {
            float ior = 1.1, eta = (inside) ? ior : 1 / ior; // are we inside or outside the surface?
            float cosi = -1 * dot(nhit, raydir);
            float k = 1 - eta * eta * (1 - cosi * cosi);
            float3 refrdir = raydir * eta + nhit * (eta *  cosi - sqrtf(k));
            refrdir = normalize(refrdir);
            reflOrig = phit - nhit * bias;
            refraction = Trace(reflOrig, refrdir, objects, objSize, depth + 1);
        }
        // the result is a mix of reflection and refraction (if the sphere is transparent)
        surfaceColor = (
            reflection * fresneleffect +
            refraction * (1 - fresneleffect) * object->transparency) * object->surfaceColor;
    }
    else {
        // it's a diffuse object, no need to raytrace any further
        for (unsigned i = 0; i < objSize; ++i) {
            if (objects[i].emissionColor.x > 0) {
                // this is a light
                float3 transmission = make_float3(1,1,1);
                float3 lightDirection = -1 * (phit - objects[i].center);
                lightDirection = normalize(lightDirection);
                for (unsigned j = 0; j < objSize; ++j) {
                    if (i != j) {
                        float t0, t1;
                        auto orig = phit + nhit * bias;
                        if (objects[j].Intersect(orig, lightDirection, t0, t1)) {
                            transmission = make_float3(0,0,0);
                            break;
                        }
                    }
                }
                surfaceColor += object->surfaceColor * transmission *
                fmaxf(float(0), dot(nhit, lightDirection)) * objects[i].emissionColor;
            }
        }
    }
    
    return surfaceColor + object->emissionColor;
}

__global__
void Render(float3* image, CudaSphere* objects, int objectSize,
            unsigned int width, unsigned int height)
{
    float invWidth = 1 / float(width), invHeight = 1 / float(height);
    float fov = 30, aspectRatio = width / float(height);
    float angle = tanf(M_PI * 0.5 * fov / 180.);

    // // Single GPU Thread
    // float3* pixel = image;
    // for (unsigned y = 0; y < height; ++y) {
    //     for (unsigned x = 0; x < width; ++x, ++pixel) {
    //         float xx = (2 * ((x + 0.5) * invWidth) - 1) * angle * aspectRatio;
    //         float yy = (1 - 2 * ((y + 0.5) * invHeight)) * angle;
    //         float3 raydir = make_float3(xx, yy, -1);
    //         raydir = normalize(raydir);
    //         float3 zero = make_float3(0,0,0);
    //         *pixel = Trace(zero, raydir, objects, objectSize, 0);
    //     }
    // }

    // Parallelization
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;   
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int i = x + width * y;

    // Trace rays
    float xx = (2 * ((x + 0.5f) * invWidth) - 1) * angle * aspectRatio; 
    float yy = (1 - 2 * ((y + 0.5f) * invHeight)) * angle; 
    float3 raydir = make_float3(xx, yy, -1); 
    raydir = normalize(raydir);
    float3 zero = make_float3(0,0,0);
    image[i] = Trace(zero, raydir, objects, objectSize, 0);
}

void CudaRT::RenderWrapper(float3* image, unsigned width, unsigned height)
{
    float3* output;    // pointer to memory for image on the device (GPU VRAM)
    checkCudaErrors(hipMalloc(&output, width*height*sizeof(float3)));

    CudaSphere* spheres;
    int size = 6;
    checkCudaErrors(hipMallocManaged(&spheres, size*sizeof(CudaSphere)));
    spheres[0] = CudaSphere(make_float3(0, -10004, -20), 10000, make_float3(0.2, 0.2, 0.2), 0, 0);
    spheres[1] = CudaSphere(make_float3(0, 0, -20), 4.0, make_float3(1.0, 0.32, 0.36), 1, 0.5);
    spheres[2] = CudaSphere(make_float3(5, -1, -15), 2, make_float3(0.9, 0.76, 0.46), 1, 0.0);
    spheres[3] = CudaSphere(make_float3(5, 0, -25), 3, make_float3(0.65, 0.77, 0.97), 1, 0.0);
    spheres[4] = CudaSphere(make_float3(-5.5, 0, -15), 3, make_float3(0.9, 0.9, 0.9), 1, 0.0);
    spheres[5] = CudaSphere(make_float3(0.0, 20, -30), 3, make_float3(0, 0, 0), 0, 0.0f, make_float3(3,3,3));

    std::cout << "Memory allocated" << std::endl;
            
    // dim3 is CUDA specific type, block and grid are required to schedule CUDA threads over streaming multiprocessors
    dim3 block(8, 8, 1);   
    dim3 grid(width / block.x, height / block.y, 1);
    
    // schedule threads on device and launch CUDA kernel from host
    Render<<< grid, block >>>(output, spheres, size, width, height);
    
    // Single GPU Thread
    // Render<<< 1, 1 >>>(output, spheres, size, width, height);

    // Wait to synchronize
    checkCudaErrors(hipDeviceSynchronize());
    std::cout << "Finish synchronization" << std::endl;

    // Copy results back
    hipMemcpy(image, output, width*height*sizeof(float3), hipMemcpyDeviceToHost);

    // free CUDA memory
    checkCudaErrors(hipFree(output));
    checkCudaErrors(hipFree(spheres));
}
